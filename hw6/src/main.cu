#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <memory>
#include <string>

#include <hip/hip_runtime.h>
#include <helper_timer.h>

#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "util.hpp"

#include "superpixel_gslic.cuh"

using namespace std;
using namespace cv;

const char *ARG0 = "homework6";

static void help_and_exit(int code = 0)
{
    ostream &os = code ? cerr : cout;
    os << "Usage: " << ARG0 << " NUM_CLUSTERS INFILE OUTFILE" << endl;
    os << endl;
    os << "\
DESCRIPTION\n\
  Run a GPU implementation of the SLIC Superpixel algorithm on an input\n\
  image. The output image will contain NUM_CLUSTERS clusters.\n\
\n\
ARGUMENTS\n\
  NUM_CLUSTERS  The number of superpixel clusters to create in the output\n\
                image. This number must be a positive integer and a perfect\n\
                square.\n\
  INFILE        The path of the input image.\n\
  OUTFILE       The path to use for the output image file.\n\
";
    exit(code);
}

#define ERR(...) do { \
    fprintf(stderr, "error: "); \
    fprintf(stderr, __VA_ARGS__); \
    fprintf(stderr, "\n"); \
    if (ret) \
        fprintf(stderr, "  %s\n", hipGetErrorString(ret)); \
    goto err; \
} while(0)

#define MARK_TIME(__label) do { \
    double __t_ms = timer.getTime(); \
    printf("%s: %.4f ms\n", (__label), __t_ms); \
    timer.reset(); \
} while (0)

int main(int argc, char *argv[])
{
    int code = 0;

    /**
     * Global initialization
     */

    ARG0 = argv[0];

    /**
     * Argument parsing
     */

    if (argc != 4)
        help_and_exit(1);

    // Parse & validate the number of clusters
    const char *n_seeds_str = argv[1];
    int n_seeds = 256;
    if (!cstr_to_int(n_seeds_str, &n_seeds))
        die("invalid number of superpixel clusters: %s", n_seeds_str);
    if (!is_perfect_square(n_seeds))
        die("invalid number of superpixel clusters, not a perfect square: %d",
            n_seeds);

    // Validate infile
    const char *infile = argv[2];
    if (!file_exists(infile))
        die("no such file: %s", infile);

    // Validate outfile
    const char *outfile = argv[3];
    if (string(outfile).rfind(".png") == string::npos)
        die("outfile must end with .png: %s", outfile);

    cout << "clusters: " << n_seeds << endl;
    cout << "infile: " << infile << endl;
    cout << "outfile: " << outfile << endl;

    /**
     * Load image, setup memory
     */

    StopWatchLinux timer;

    timer.start();
    Mat img_rgb = imread(infile, IMREAD_COLOR);
    timer.stop();
    MARK_TIME("image load time");

    unsigned int width = img_rgb.cols;
    unsigned int height = img_rgb.rows;
    unsigned int n_pixels = width * height;

    // Convert to CIELAB
    Mat h_img_lab;
    cvtColor(img_rgb, h_img_lab, COLOR_BGR2Lab);

    hipError_t ret = hipSuccess;

    Pixel_t *d_img = NULL;
    size_t img_lab_size = sizeof(Pixel_t) * n_pixels;

    Seed_t *d_seeds = NULL;
    size_t seeds_size = sizeof(Seed_t) * n_seeds;

    ClosestSeed_t *d_distances = NULL;
    size_t distances_size = sizeof(ClosestSeed_t) * n_pixels;

    Seed_t *d_seed_sigmas = NULL;
    size_t seed_sigmas_size = sizeof(Seed_t) * n_seeds;

    double *d_seed_pixel_counts = NULL;
    size_t seed_pixel_counts_size = sizeof(double) * n_seeds;

    if ((ret = hipMalloc(&d_img, img_lab_size)))
        ERR("failed to allocate space for image on device");

    if ((ret = hipMalloc(&d_seeds, seeds_size)))
        ERR("failed to allocate space for seeds array on device");

    if ((ret = hipMalloc(&d_distances, distances_size)))
        ERR("failed to allocate space for distances array on device");

    if ((ret = hipMalloc(&d_seed_sigmas, seed_sigmas_size)))
        ERR("failed to allocate space for seed sigmas array on device");

    if ((ret = hipMalloc(&d_seed_pixel_counts, seed_pixel_counts_size)))
        ERR("failed to allocate space for seed sizes array on device");

    /**
     * Copy to kernel, run, copy back
     */

    timer.start();

    if ((ret = hipMemcpy(d_img, h_img_lab.data, img_lab_size,
                          hipMemcpyHostToDevice)))
        ERR("failed to copy image to device");

    superpixel_gslic
        <<< 1, 1 >>>
    (
        d_img, width, height,
        d_seeds, n_seeds,
        d_distances, d_seed_sigmas, d_seed_pixel_counts
    );

    if ((ret = hipMemcpy(h_img_lab.data, d_img, img_lab_size,
                          hipMemcpyDeviceToHost)))
        ERR("failed to copy output image to host");

    timer.stop();
    MARK_TIME("kernel latency");

    /**
     * Write the output image to filesystem
     */

    // Convert back to RGB
    cvtColor(h_img_lab, img_rgb, COLOR_Lab2BGR);

    if (!imwrite(outfile, img_rgb))
        ERR("failed to write output image to file: %s", outfile);

    /**
     * Cleanup
     */

    goto end;

err:
    code = 1;

end:
    hipFree(d_seed_pixel_counts);
    hipFree(d_seed_sigmas);
    hipFree(d_distances);
    hipFree(d_seeds);
    hipFree(d_img);

    return code;
}
